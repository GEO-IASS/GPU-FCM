/*=======================================================================
 *
 * load.c
 * A file of loading routines, MRI, ATR and a simple test case.
 *
 * Note: ATR support is by default included, disable by undef'ing
 *  ATR_SUPPORT (a stub will be used instead)
 *
 * $Id: load.c,v 1.3 2002/07/12 20:48:48 eschrich Exp $
 * Steven Eschrich
 *
 * Copyright (C) 2002 University of South Florida
 *
 * This program is free software; you can redistribute it and/or modify it
 * under the terms of the GNU General Public License as published by the
 * Free Software Foundation; either version 2 of the License, or (at
 * your option) any later version.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with this program; if not, write to the Free Software Foundation, Inc.,
 * 59 Temple Place, Suite 330, Boston, MA 02111-1307 USA
 *=======================================================================*/

#define ATR_SUPPORT
#define DIM 256

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#ifndef HANDLE_ERROR
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
#endif
#ifdef ATR_SUPPORT
 #include <tiffio.h>
#endif

#include "utils.h"


/* Load the dummy test file into the dataset. The information is
   produced without using a file, so this is generated in-memory

   The variable S sets the dimension, the variable N sets the number
   of examples.
*/
int load_test_data(float **ds,float **ds_d, int *s, int *n)
{
  int i;
  float *X;
  int S=1;
  int N=500;
  fprintf(stdout,"Loading test dataset...");
  X=(float *)CALLOC(N*S,sizeof(float));
  hipMalloc(ds_d,N*S*sizeof(float));
  for (i=0; i < N; i++) {
    if ( i < 100 ) X[i*S+0]=i;
    else if ( i < 200 ) X[i*S+0]=i-100;
    else if ( i < 300 ) X[i*S+0]=i-200;
    else if ( i < 400 ) X[i*S+0]=255-i-300;
    else if ( i < 500 ) X[i*S+0]=255-i-400;
  }
  *n=N;
  *s=S;
  *ds=X;
  fprintf(stdout,"done (%d exmaples).\n", N);
  hipMemcpy(*ds_d,X,N*S* sizeof(float),hipMemcpyHostToDevice);

  return 0;
}



/*

  loadMRI()

  Loads an MRI image into memory as an int ** array of feature
  values.
*/
int load_mri_data(char *filename, float **ds,float **ds_d, int *s, int *n)
{
  FILE *fp;
  int image_length=DIM;
  int image_width=DIM;
  int S=3;
  int i,j;
  unsigned short int *buf;
  float *X;
  fprintf(stderr,"Loading MRI image %s...", filename);

  fp=FOPEN(filename,"r");

  /* Allocate storage */
  X=(float *)CALLOC(image_length * image_width * S,sizeof(float));
  HANDLE_ERROR(hipMalloc(ds_d,image_length * image_width * S*sizeof(float)));
  buf=(unsigned short int *)CALLOC(image_width*image_length, sizeof(unsigned short int));
  for (i=0; i < S; i++) {
    fread(buf,2,image_width*image_length,fp);
    for (j=0; j < image_width*image_length;j++) {
      X[j*S+i]=buf[j];
    }
  }

  fclose(fp);
  fprintf(stderr,"done (%d examples).\n", image_width * image_length);
  *ds=X;
  *s=S;
  *n=image_length*image_width;
  hipGetErrorString(hipMemcpy(*ds_d,X,image_length * image_width * S * sizeof(float),hipMemcpyHostToDevice));

  return 0;

}



/*

  load_atr_data()

  Loads an ATR TIFF image into memory as an int ** array of feature
  values.
*/

#ifdef ATR_SUPPORT
int kernel[5][5]={
{1,-2,0,2,-1},
{0,0,0,0,0},
{-2,4,0,-4,1},
{0,0,0,0,0},
{1,-2,0,2,-1}
};


int load_atr_data(char *filename, float **ds,float **ds_d ,int *s, int *n)
{
  TIFF *fp;
  int image_length, image_width;
  int i,j,rc,cc;
  unsigned char *buf;
  float *X;
  float val;
  fprintf(stderr,"Loading ATR image %s...", filename);

  if ( (fp=TIFFOpen(filename,"r")) == 0 )
    die("Can't open %s", filename);
  TIFFGetField(fp, TIFFTAG_IMAGELENGTH, &image_length);
  TIFFGetField(fp, TIFFTAG_IMAGEWIDTH, &image_width);

  /* Allocate storage */
  X=(float *)CALLOC(image_length * image_width * 2, sizeof(float));
hipMalloc(ds_d,image_length * image_width * 2* sizeof(float));
  buf=(unsigned char *)CALLOC(image_width, sizeof(unsigned char));
  for (i=0; i < image_length; i++) {
    TIFFReadScanline(fp, buf, i, 0);
    for (j=0; j < image_width;j++) {
      X[i*image_width + j * image_length +0]=buf[j];
    }
  }

  TIFFClose(fp);
  fprintf(stderr,"done (%d examples).\n", image_width * image_length);

  /* normalize(); */

  /* If more than one feature asked for, generate Laws' texture values */
  fprintf(stderr,"Generating Laws' texture feature...");
  for (i=0; i < image_length; i++) {
    for (j=0; j < image_width;j++) {
      val=0;
           for (rc=0; rc < 5; rc++ ) {
              for (cc=0; cc < 5;cc++) {
                 if (i-2+rc < 0 || i-2+rc >= image_length ||
	               j-2+cc < 0 || j-2+cc >= image_width)
                    continue;
                 else
                    val += X[(i-2+rc)*(image_width) + (j-2+cc)*image_length+0] * (float)kernel[rc][cc];
              }
	   }
	   X[i*image_width + j * image_length +1]=val;
	 }
      }

  fprintf(stderr,"done.\n");
  *s=2;
  *n=image_length*image_width;
  *ds=X;

  hipMemcpy(*ds_d,X,image_length * image_width * 2 * sizeof(float),hipMemcpyHostToDevice);
  return 0;

}



#else
int load_atr_data()
{
  die("ATR support (TIFF support) not compiled in.\n");
}
#endif
